#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_3_stateupdater_codeobject.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

////// SUPPORT CODE ///////
namespace {
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }


        
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                inline __device__ int _brian_atomicAdd(int* address, int val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ float _brian_atomicAdd(float* address, float val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ double _brian_atomicAdd(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val +
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicMul(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val * assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicMul(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val *
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicMul(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val *
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicDiv(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val / assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicDiv(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val /
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicDiv(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val /
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }

}




__global__ void
kernel_neurongroup_3_stateupdater_codeobject(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_neurongroup_3_s_AMPA,
	double* _ptr_array_neurongroup_3_V,
	const double _value_array_defaultclock_dt,
	double* _ptr_array_neurongroup_3_s_AHP_BS
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _nums_AMPA = 10;
	const int _numV = 10;
	const int _nums_AHP_BS = 10;

    ///// kernel_lines /////
        
    const double* _ptr_array_defaultclock_dt = &_value_array_defaultclock_dt;


    assert(THREADS_PER_BLOCK == blockDim.x);


    if(_idx >= _N)
    {
        return;
    }


    ///// scalar_code /////
        
    const double dt = _ptr_array_defaultclock_dt[0];
    const double _lio_1 = 1.0f*(- dt)/0.0083;
    const double _lio_2 = 1.0f*dt/1.0699999999999999e-10;
    const double _lio_3 = - 2.32e-09;
    const double _lio_4 = - (-0.068);
    const double _lio_5 = - (-0.07);
    const double _lio_6 = - 0.0;
    const double _lio_7 = 1.0f*(- dt)/0.0025;


    {
        ///// vector_code /////
                
        double s_AMPA = _ptr_array_neurongroup_3_s_AMPA[_idx];
        double s_AHP_BS = _ptr_array_neurongroup_3_s_AHP_BS[_idx];
        double V = _ptr_array_neurongroup_3_V[_idx];
        const double _s_AMPA = (_lio_1 * s_AMPA) + s_AMPA;
        const double _V = V + (_lio_2 * ((_lio_3 * (_lio_4 + V)) - ((1.0000000000000002e-10 * (s_AHP_BS * (_lio_5 + V))) + (7e-10 * (s_AMPA * (_lio_6 + V))))));
        const double _s_AHP_BS = (_lio_7 * s_AHP_BS) + s_AHP_BS;
        s_AMPA = _s_AMPA;
        V = _V;
        s_AHP_BS = _s_AHP_BS;
        _ptr_array_neurongroup_3_s_AMPA[_idx] = s_AMPA;
        _ptr_array_neurongroup_3_s_AHP_BS[_idx] = s_AHP_BS;
        _ptr_array_neurongroup_3_V[_idx] = V;


    }
}

void _run_neurongroup_3_stateupdater_codeobject()
{
    using namespace brian;


    const int _N = 10;

    ///// HOST_CONSTANTS ///////////
    const int _nums_AMPA = 10;
		const int _numV = 10;
		const int _nums_AHP_BS = 10;


    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
        // get number of blocks and threads
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    kernel_neurongroup_3_stateupdater_codeobject, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;

        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_neurongroup_3_stateupdater_codeobject, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_neurongroup_3_stateupdater_codeobject))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_neurongroup_3_stateupdater_codeobject "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_neurongroup_3_stateupdater_codeobject, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }
        else
        {
            printf("INFO kernel_neurongroup_3_stateupdater_codeobject\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }


    kernel_neurongroup_3_stateupdater_codeobject<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            dev_array_neurongroup_3_s_AMPA,
			dev_array_neurongroup_3_V,
			_array_defaultclock_dt[0],
			dev_array_neurongroup_3_s_AHP_BS
        );

    CUDA_CHECK_ERROR("kernel_neurongroup_3_stateupdater_codeobject");


}


