#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_1_stateupdater_codeobject_1.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

////// SUPPORT CODE ///////
namespace {
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }


        
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                inline __device__ int _brian_atomicAdd(int* address, int val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ float _brian_atomicAdd(float* address, float val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ double _brian_atomicAdd(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val +
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicMul(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val * assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicMul(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val *
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicMul(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val *
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicDiv(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val / assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicDiv(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val /
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicDiv(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val /
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }

}




__global__ void
kernel_neurongroup_1_stateupdater_codeobject_1(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_neurongroup_1_s_AMPA,
	double* _ptr_array_neurongroup_1_s_ahp_GO,
	double* _ptr_array_neurongroup_1_s_NMDA_2,
	double* _ptr_array_neurongroup_1_s_NMDA_1,
	double* _ptr_array_neurongroup_1_V,
	const double _value_array_defaultclock_dt
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _nums_AMPA = 10;
	const int _nums_ahp_GO = 10;
	const int _nums_NMDA_2 = 10;
	const int _nums_NMDA_1 = 10;
	const int _numV = 10;

    ///// kernel_lines /////
        
    const double* _ptr_array_defaultclock_dt = &_value_array_defaultclock_dt;


    assert(THREADS_PER_BLOCK == blockDim.x);


    if(_idx >= _N)
    {
        return;
    }


    ///// scalar_code /////
        
    const double dt = _ptr_array_defaultclock_dt[0];
    const double _lio_1 = 1.0f*(- dt)/0.0015;
    const double _lio_2 = 1.0f*(- dt)/0.031;
    const double _lio_3 = 1.0f*(- dt)/0.17;
    const double _lio_4 = 1.0f*(- dt)/0.005;
    const double _lio_5 = 1.0f*dt/2.8e-11;
    const double _lio_6 = - 2.3e-09;
    const double _lio_7 = - (-0.055);
    const double _lio_8 = - 0.0;
    const double _lio_9 = 0.33 * 3.0000000000000004e-08;
    const double _lio_10 = - 0.0;
    const double _lio_11 = 0.67 * 3.0000000000000004e-08;
    const double _lio_12 = - (-0.0727);


    {
        ///// vector_code /////
                
        double s_ahp_GO = _ptr_array_neurongroup_1_s_ahp_GO[_idx];
        double s_NMDA_1 = _ptr_array_neurongroup_1_s_NMDA_1[_idx];
        double s_AMPA = _ptr_array_neurongroup_1_s_AMPA[_idx];
        double s_NMDA_2 = _ptr_array_neurongroup_1_s_NMDA_2[_idx];
        double V = _ptr_array_neurongroup_1_V[_idx];
        const double _s_AMPA = (_lio_1 * s_AMPA) + s_AMPA;
        const double _s_NMDA_1 = (_lio_2 * s_NMDA_1) + s_NMDA_1;
        const double _s_NMDA_2 = (_lio_3 * s_NMDA_2) + s_NMDA_2;
        const double _s_ahp_GO = (_lio_4 * s_ahp_GO) + s_ahp_GO;
        const double _V = V + (_lio_5 * ((_lio_6 * (_lio_7 + V)) - ((((1.8e-10 * (s_AMPA * (_lio_8 + V))) + (_lio_9 * (s_NMDA_1 * (_lio_10 + V)))) + (_lio_11 * (s_NMDA_2 * (_lio_8 + V)))) + (2e-08 * (s_ahp_GO * (_lio_12 + V))))));
        s_AMPA = _s_AMPA;
        s_NMDA_1 = _s_NMDA_1;
        s_NMDA_2 = _s_NMDA_2;
        s_ahp_GO = _s_ahp_GO;
        V = _V;
        _ptr_array_neurongroup_1_s_ahp_GO[_idx] = s_ahp_GO;
        _ptr_array_neurongroup_1_s_NMDA_1[_idx] = s_NMDA_1;
        _ptr_array_neurongroup_1_s_AMPA[_idx] = s_AMPA;
        _ptr_array_neurongroup_1_s_NMDA_2[_idx] = s_NMDA_2;
        _ptr_array_neurongroup_1_V[_idx] = V;


    }
}

void _run_neurongroup_1_stateupdater_codeobject_1()
{
    using namespace brian;

    const std::clock_t _start_time = std::clock();

    const int _N = 10;

    ///// HOST_CONSTANTS ///////////
    const int _nums_AMPA = 10;
		const int _nums_ahp_GO = 10;
		const int _nums_NMDA_2 = 10;
		const int _nums_NMDA_1 = 10;
		const int _numV = 10;


    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
        // get number of blocks and threads
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    kernel_neurongroup_1_stateupdater_codeobject_1, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;

        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_neurongroup_1_stateupdater_codeobject_1, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_neurongroup_1_stateupdater_codeobject_1))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_neurongroup_1_stateupdater_codeobject_1 "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_neurongroup_1_stateupdater_codeobject_1, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }
        else
        {
            printf("INFO kernel_neurongroup_1_stateupdater_codeobject_1\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }


    kernel_neurongroup_1_stateupdater_codeobject_1<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            dev_array_neurongroup_1_s_AMPA,
			dev_array_neurongroup_1_s_ahp_GO,
			dev_array_neurongroup_1_s_NMDA_2,
			dev_array_neurongroup_1_s_NMDA_1,
			dev_array_neurongroup_1_V,
			_array_defaultclock_dt[0]
        );

    CUDA_CHECK_ERROR("kernel_neurongroup_1_stateupdater_codeobject_1");


    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );
    const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    neurongroup_1_stateupdater_codeobject_1_profiling_info += _run_time;
}


