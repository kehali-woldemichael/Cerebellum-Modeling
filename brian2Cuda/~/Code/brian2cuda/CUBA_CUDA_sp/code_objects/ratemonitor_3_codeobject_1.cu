#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/ratemonitor_3_codeobject_1.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

////// SUPPORT CODE ///////
namespace {
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }


        
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                inline __device__ int _brian_atomicAdd(int* address, int val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ float _brian_atomicAdd(float* address, float val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ double _brian_atomicAdd(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val +
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicMul(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val * assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicMul(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val *
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicMul(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val *
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicDiv(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val / assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicDiv(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val /
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicDiv(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val /
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }

}




__global__ void
kernel_ratemonitor_3_codeobject_1(
    int32_t current_iteration,
    double* ratemonitor_rate,
    double* ratemonitor_t,
    ///// KERNEL_PARAMETERS /////
    const double _value_array_defaultclock_t,
	int32_t* _ptr_array_neurongroup_1__spikespace,
	double* _ptr_array_ratemonitor_3_rate,
	const int _numrate,
	const double _value_array_defaultclock_dt,
	double* _ptr_array_ratemonitor_3_t,
	const int _numt
    )
{
    using namespace brian;

    ///// KERNEL_CONSTANTS /////
    const int _num_spikespace = 11;

    ///// kernel_lines /////
        
    const double* _ptr_array_defaultclock_t = &_value_array_defaultclock_t;
    const double* _ptr_array_defaultclock_dt = &_value_array_defaultclock_dt;


    int num_spikes = 0;

    if (_num_spikespace-1 != 10)  // we have a subgroup
    {
        // TODO shouldn't this be 'i < _num_spikespace -1'?
        for (int i=0; i < _num_spikespace; i++)
        {
            const int spiking_neuron = _ptr_array_neurongroup_1__spikespace[i];
            if (spiking_neuron != -1)
            {
                // check if spiking neuron is in this subgroup
                if (0 <= spiking_neuron && spiking_neuron < 10)
                    num_spikes++;
            }
            else  // end of spiking neurons
            {
                break;
            }
        }
    }
    else  // we don't have a subgroup
    {
        num_spikes = _ptr_array_neurongroup_1__spikespace[10];
    }

    // TODO: we should be able to use _ptr_array_ratemonitor_3_rate and _ptr_array_ratemonitor_3_t here instead of passing these
    //       additional pointers. But this results in thrust::system_error illegal memory access.
    //       Don't know why... _ptr_array_ratemonitor_3_rate and ratemonitor_rate should be the same...
    ratemonitor_rate[current_iteration] = 1.0*num_spikes/_ptr_array_defaultclock_dt[0]/10;
    ratemonitor_t[current_iteration] = _ptr_array_defaultclock_t[0];
}

void _run_ratemonitor_3_codeobject_1()
{
    using namespace brian;

    const std::clock_t _start_time = std::clock();


    ///// HOST_CONSTANTS ///////////
    const int _num_spikespace = 11;
		double* const _array_ratemonitor_3_rate = thrust::raw_pointer_cast(&dev_dynamic_array_ratemonitor_3_rate[0]);
		const int _numrate = dev_dynamic_array_ratemonitor_3_rate.size();
		double* const _array_ratemonitor_3_t = thrust::raw_pointer_cast(&dev_dynamic_array_ratemonitor_3_t[0]);
		const int _numt = dev_dynamic_array_ratemonitor_3_t.size();

int current_iteration = defaultclock.timestep[0];
static int start_offset = current_iteration;

    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
int num_iterations = defaultclock.i_end;
int size_till_now = dev_dynamic_array_ratemonitor_3_t.size();
THRUST_CHECK_ERROR(
        dev_dynamic_array_ratemonitor_3_t.resize(num_iterations + size_till_now - start_offset)
        );
THRUST_CHECK_ERROR(
        dev_dynamic_array_ratemonitor_3_rate.resize(num_iterations + size_till_now - start_offset)
        );
num_threads = 1;
num_blocks = 1;

        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_ratemonitor_3_codeobject_1, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_ratemonitor_3_codeobject_1))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_ratemonitor_3_codeobject_1 "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_ratemonitor_3_codeobject_1, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }
        else
        {
            printf("INFO kernel_ratemonitor_3_codeobject_1\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }


kernel_ratemonitor_3_codeobject_1<<<num_blocks, num_threads>>>(
    current_iteration - start_offset,
    thrust::raw_pointer_cast(&(dev_dynamic_array_ratemonitor_3_rate[0])),
    thrust::raw_pointer_cast(&(dev_dynamic_array_ratemonitor_3_t[0])),
    ///// HOST_PARAMETERS /////
    _array_defaultclock_t[0],
			dev_array_neurongroup_1__spikespace[current_idx_array_neurongroup_1__spikespace],
			_array_ratemonitor_3_rate,
			_numrate,
			_array_defaultclock_dt[0],
			_array_ratemonitor_3_t,
			_numt);

CUDA_CHECK_ERROR("kernel_ratemonitor_3_codeobject_1");


    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );
    const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    ratemonitor_3_codeobject_1_profiling_info += _run_time;
}


