#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_3_thresholder_codeobject_1.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

////// SUPPORT CODE ///////
namespace {
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }

        __global__ void
        _reset_neurongroup_3_thresholder_codeobject_1(
            int32_t* eventspace
            )
        {
            using namespace brian;

            int _idx = blockIdx.x * blockDim.x + threadIdx.x;

            if (_idx == 0)
            {
                // reset eventspace counter
                eventspace[10] = 0;
            }

            if (_idx < 10)
            {
                // reset eventspace
                eventspace[_idx] = -1;
            }
        }

        
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                inline __device__ int _brian_atomicAdd(int* address, int val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ float _brian_atomicAdd(float* address, float val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ double _brian_atomicAdd(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val +
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicMul(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val * assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicMul(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val *
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicMul(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val *
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicDiv(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val / assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicDiv(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val /
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicDiv(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val /
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }

}




__global__ void
kernel_neurongroup_3_thresholder_codeobject_1(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_neurongroup_3_V,
	int32_t* _ptr_array_neurongroup_3__spikespace
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _numV = 10;
	const int _num_spikespace = 11;

    ///// kernel_lines /////
        


    assert(THREADS_PER_BLOCK == blockDim.x);


    if(_idx >= _N)
    {
        return;
    }


    ///// scalar_code /////
        



    {// there might be the same variable defined in scalar and vector code
    ///// vector_code /////
        
    const double V = _ptr_array_neurongroup_3_V[_idx];
    const char _cond = V > (-0.055);


    if (_cond)
    {
        int32_t spike_index = atomicAdd(&_ptr_array_neurongroup_3__spikespace[_N], 1);
        _ptr_array_neurongroup_3__spikespace[spike_index] = _idx;
    }
    }
}

void _run_neurongroup_3_thresholder_codeobject_1()
{
    using namespace brian;

    const std::clock_t _start_time = std::clock();

    const int _N = 10;

    ///// HOST_CONSTANTS ///////////
    const int _numV = 10;
		const int _num_spikespace = 11;


    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
        // get number of blocks and threads
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    kernel_neurongroup_3_thresholder_codeobject_1, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;

        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_neurongroup_3_thresholder_codeobject_1, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_neurongroup_3_thresholder_codeobject_1))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_neurongroup_3_thresholder_codeobject_1 "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_neurongroup_3_thresholder_codeobject_1, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }
        else
        {
            printf("INFO kernel_neurongroup_3_thresholder_codeobject_1\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }

        _reset_neurongroup_3_thresholder_codeobject_1<<<num_blocks, num_threads>>>(
                dev_array_neurongroup_3__spikespace[current_idx_array_neurongroup_3__spikespace]
            );

        CUDA_CHECK_ERROR("_reset_neurongroup_3_thresholder_codeobject_1");

    kernel_neurongroup_3_thresholder_codeobject_1<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            dev_array_neurongroup_3_V,
			dev_array_neurongroup_3__spikespace[current_idx_array_neurongroup_3__spikespace]
        );

    CUDA_CHECK_ERROR("kernel_neurongroup_3_thresholder_codeobject_1");


    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );
    const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    neurongroup_3_thresholder_codeobject_1_profiling_info += _run_time;
}


