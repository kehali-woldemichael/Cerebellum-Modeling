#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/poissongroup_thresholder_codeobject_1.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

////// SUPPORT CODE ///////
namespace {
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }

        __global__ void
        _reset_poissongroup_thresholder_codeobject_1(
            int32_t* eventspace
            )
        {
            using namespace brian;

            int _idx = blockIdx.x * blockDim.x + threadIdx.x;

            if (_idx == 0)
            {
                // reset eventspace counter
                eventspace[100] = 0;
            }

            if (_idx < 100)
            {
                // reset eventspace
                eventspace[_idx] = -1;
            }
        }

        
    #define _rand(vectorisation_idx) (_ptr_array_poissongroup_thresholder_codeobject_1_rand[vectorisation_idx])
    template <typename T>
    __host__ __device__
    double _brian_cos(T value)
    {
    #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
        return cos((double)value);
    #else
        return cos(value);
    #endif
    }
    inline __host__ __device__
    float _brian_cos(float value)
    {
        return cos(value);
    }
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                inline __device__ int _brian_atomicAdd(int* address, int val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ float _brian_atomicAdd(float* address, float val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ double _brian_atomicAdd(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val +
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicMul(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val * assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicMul(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val *
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicMul(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val *
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicDiv(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val / assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicDiv(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val /
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicDiv(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val /
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }

}




__global__ void
kernel_poissongroup_thresholder_codeobject_1(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    int32_t* _ptr_array_poissongroup__spikespace,
	const double _value_array_defaultclock_t,
	const double _value_array_defaultclock_dt,
	double* _ptr_array_poissongroup_thresholder_codeobject_1_rand
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _num_spikespace = 101;

    ///// kernel_lines /////
        
    const double* _ptr_array_defaultclock_t = &_value_array_defaultclock_t;
    const double* _ptr_array_defaultclock_dt = &_value_array_defaultclock_dt;


    assert(THREADS_PER_BLOCK == blockDim.x);


    if(_idx >= _N)
    {
        return;
    }


    ///// scalar_code /////
        
    const double t = _ptr_array_defaultclock_t[0];
    const double _lio_1 = (22.5 + (7.5 * _brian_cos((2.0 * ((3.141592653589793 * t) * 0.5)) + 3.141592653589793))) * 1.0;



    {// there might be the same variable defined in scalar and vector code
    ///// vector_code /////
        
    const double dt = _ptr_array_defaultclock_dt[0];
    const double rates = _lio_1;
    const char _cond = _rand(_vectorisation_idx + 0 * _N) < (dt * rates);


    if (_cond)
    {
        int32_t spike_index = atomicAdd(&_ptr_array_poissongroup__spikespace[_N], 1);
        _ptr_array_poissongroup__spikespace[spike_index] = _idx;
    }
    }
}

void _run_poissongroup_thresholder_codeobject_1()
{
    using namespace brian;

    const std::clock_t _start_time = std::clock();

    const int _N = 100;

    ///// HOST_CONSTANTS ///////////
    const int _num_spikespace = 101;


    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
        // get number of blocks and threads
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    kernel_poissongroup_thresholder_codeobject_1, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;

        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_poissongroup_thresholder_codeobject_1, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_poissongroup_thresholder_codeobject_1))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_poissongroup_thresholder_codeobject_1 "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_poissongroup_thresholder_codeobject_1, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }
        else
        {
            printf("INFO kernel_poissongroup_thresholder_codeobject_1\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }

        _reset_poissongroup_thresholder_codeobject_1<<<num_blocks, num_threads>>>(
                dev_array_poissongroup__spikespace[current_idx_array_poissongroup__spikespace]
            );

        CUDA_CHECK_ERROR("_reset_poissongroup_thresholder_codeobject_1");

    kernel_poissongroup_thresholder_codeobject_1<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            dev_array_poissongroup__spikespace[current_idx_array_poissongroup__spikespace],
			_array_defaultclock_t[0],
			_array_defaultclock_dt[0],
			dev_poissongroup_thresholder_codeobject_1_rand
        );

    CUDA_CHECK_ERROR("kernel_poissongroup_thresholder_codeobject_1");


    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );
    const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    poissongroup_thresholder_codeobject_1_profiling_info += _run_time;
}


