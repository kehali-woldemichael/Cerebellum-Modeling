#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_stateupdater_codeobject.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

////// SUPPORT CODE ///////
namespace {
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }


        
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                inline __device__ int _brian_atomicAdd(int* address, int val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ float _brian_atomicAdd(float* address, float val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ double _brian_atomicAdd(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val +
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicMul(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val * assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicMul(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val *
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicMul(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val *
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicDiv(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val / assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicDiv(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val /
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicDiv(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val /
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }

}




__global__ void
kernel_neurongroup_stateupdater_codeobject(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_neurongroup_s_GABA_1,
	double* _ptr_array_neurongroup_s_GABA_2,
	double* _ptr_array_neurongroup_s_AMPA,
	double* _ptr_array_neurongroup_V,
	const double _value_array_defaultclock_dt,
	double* _ptr_array_neurongroup_s_NMDA,
	double* _ptr_array_neurongroup_s_ahp_GR
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _nums_GABA_1 = 100;
	const int _nums_GABA_2 = 100;
	const int _nums_AMPA = 100;
	const int _numV = 100;
	const int _nums_NMDA = 100;
	const int _nums_ahp_GR = 100;

    ///// kernel_lines /////
        
    const double* _ptr_array_defaultclock_dt = &_value_array_defaultclock_dt;


    assert(THREADS_PER_BLOCK == blockDim.x);


    if(_idx >= _N)
    {
        return;
    }


    ///// scalar_code /////
        
    const double dt = _ptr_array_defaultclock_dt[0];
    const double _lio_1 = 1.0f*(- dt)/0.0012;
    const double _lio_2 = 1.0f*(- dt)/0.052000000000000005;
    const double _lio_3 = 1.0f*(- dt)/0.007;
    const double _lio_4 = 1.0f*(- dt)/0.059000000000000004;
    const double _lio_5 = 1.0f*dt/3.1e-12;
    const double _lio_6 = - 4.3e-10;
    const double _lio_7 = - (-0.058);
    const double _lio_8 = - 0.0;
    const double _lio_9 = 0.43 * 2.8000000000000003e-11;
    const double _lio_10 = - (-0.082);
    const double _lio_11 = 0.57 * 2.8000000000000003e-11;
    const double _lio_12 = - (-0.082);
    const double _lio_13 = 1.0f*(- dt)/0.005;


    {
        ///// vector_code /////
                
        double s_AMPA = _ptr_array_neurongroup_s_AMPA[_idx];
        double s_NMDA = _ptr_array_neurongroup_s_NMDA[_idx];
        double s_ahp_GR = _ptr_array_neurongroup_s_ahp_GR[_idx];
        double s_GABA_2 = _ptr_array_neurongroup_s_GABA_2[_idx];
        double V = _ptr_array_neurongroup_V[_idx];
        double s_GABA_1 = _ptr_array_neurongroup_s_GABA_1[_idx];
        const double _s_AMPA = (_lio_1 * s_AMPA) + s_AMPA;
        const double _s_NMDA = (_lio_2 * s_NMDA) + s_NMDA;
        const double _s_GABA_1 = (_lio_3 * s_GABA_1) + s_GABA_1;
        const double _s_GABA_2 = (_lio_4 * s_GABA_2) + s_GABA_2;
        const double _V = V + (_lio_5 * ((_lio_6 * (_lio_7 + V)) - (((((1.8e-10 * (s_AMPA * (_lio_8 + V))) + (_lio_9 * (s_GABA_1 * (_lio_10 + V)))) + (_lio_11 * (s_GABA_2 * (_lio_10 + V)))) + (2.5000000000000004e-11 * (s_NMDA * (_lio_8 + V)))) + (1e-09 * (s_ahp_GR * (_lio_12 + V))))));
        const double _s_ahp_GR = (_lio_13 * s_ahp_GR) + s_ahp_GR;
        s_AMPA = _s_AMPA;
        s_NMDA = _s_NMDA;
        s_GABA_1 = _s_GABA_1;
        s_GABA_2 = _s_GABA_2;
        V = _V;
        s_ahp_GR = _s_ahp_GR;
        _ptr_array_neurongroup_s_AMPA[_idx] = s_AMPA;
        _ptr_array_neurongroup_s_NMDA[_idx] = s_NMDA;
        _ptr_array_neurongroup_s_ahp_GR[_idx] = s_ahp_GR;
        _ptr_array_neurongroup_s_GABA_2[_idx] = s_GABA_2;
        _ptr_array_neurongroup_V[_idx] = V;
        _ptr_array_neurongroup_s_GABA_1[_idx] = s_GABA_1;


    }
}

void _run_neurongroup_stateupdater_codeobject()
{
    using namespace brian;


    const int _N = 100;

    ///// HOST_CONSTANTS ///////////
    const int _nums_GABA_1 = 100;
		const int _nums_GABA_2 = 100;
		const int _nums_AMPA = 100;
		const int _numV = 100;
		const int _nums_NMDA = 100;
		const int _nums_ahp_GR = 100;


    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
        // get number of blocks and threads
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    kernel_neurongroup_stateupdater_codeobject, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;

        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_neurongroup_stateupdater_codeobject, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_neurongroup_stateupdater_codeobject))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_neurongroup_stateupdater_codeobject "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_neurongroup_stateupdater_codeobject, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }
        else
        {
            printf("INFO kernel_neurongroup_stateupdater_codeobject\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }


    kernel_neurongroup_stateupdater_codeobject<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            dev_array_neurongroup_s_GABA_1,
			dev_array_neurongroup_s_GABA_2,
			dev_array_neurongroup_s_AMPA,
			dev_array_neurongroup_V,
			_array_defaultclock_dt[0],
			dev_array_neurongroup_s_NMDA,
			dev_array_neurongroup_s_ahp_GR
        );

    CUDA_CHECK_ERROR("kernel_neurongroup_stateupdater_codeobject");


}


