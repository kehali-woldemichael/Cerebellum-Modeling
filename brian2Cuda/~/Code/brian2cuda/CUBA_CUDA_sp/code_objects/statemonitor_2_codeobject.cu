#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/statemonitor_2_codeobject.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

////// SUPPORT CODE ///////
namespace {
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }


        
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                inline __device__ int _brian_atomicAdd(int* address, int val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ float _brian_atomicAdd(float* address, float val)
                {
                    // hardware implementation
                    return atomicAdd(address, val);
                }
                inline __device__ double _brian_atomicAdd(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val +
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicMul(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val * assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicMul(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val *
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicMul(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val *
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }
                inline __device__ int _brian_atomicDiv(int* address, int val)
                {
                    // software implementation
                    int old = *address, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address, assumed, val / assumed);
                    } while (assumed != old);
                    return old;
                }
                inline __device__ float _brian_atomicDiv(float* address, float val)
                {
                    // software implementation
                    int* address_as_int = (int*)address;
                    int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __float_as_int(val /
                                               __int_as_float(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __int_as_float(old);
                }
                inline __device__ double _brian_atomicDiv(double* address, double val)
                {
                    // software implementation
                    unsigned long long int* address_as_int = (unsigned long long int*)address;
                    unsigned long long int old = *address_as_int, assumed;
                    do {
                        assumed = old;
                        old = atomicCAS(address_as_int, assumed,
                                        __double_as_longlong(val /
                                               __longlong_as_double(assumed)));
                    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
                    } while (assumed != old);
                    return __longlong_as_double(old);
                }

}




__global__ void
kernel_statemonitor_2_codeobject(
    int _num_indices,
    int32_t* indices,
    int current_iteration,
    double** monitor_V,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_neurongroup_3_V,
	double* _ptr_array_statemonitor_2_t,
	const int _numt,
	int32_t* _ptr_array_statemonitor_2_N
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    if(tid > _num_indices)
    {
        return;
    }
    int32_t _idx = indices[tid];

    ///// KERNEL_CONSTANTS /////
    const int _num_source_V = 10;
	const int _numN = 1;

    ///// kernel_lines /////
        


    ///// scalar_code /////
        


    // need different scope here since scalar_code and vector_code can
    // declare the same variables
    {
        ///// vector_code /////
                
        const double _source_V = _ptr_array_neurongroup_3_V[_idx];
        const double _to_record_V = _source_V;


        monitor_V[tid][current_iteration] = _to_record_V;
    }
}

void _run_statemonitor_2_codeobject()
{
    using namespace brian;



    ///// HOST_CONSTANTS ///////////
    const int _num_source_V = 10;
		double* const _array_statemonitor_2_t = thrust::raw_pointer_cast(&dev_dynamic_array_statemonitor_2_t[0]);
		const int _numt = dev_dynamic_array_statemonitor_2_t.size();
		const int _numN = 1;

// TODO: this pushes a new value to the device each time step? Looks
// inefficient, can we keep the t values on the host instead? Do we need them
// on the device?
dev_dynamic_array_statemonitor_2_t.push_back(defaultclock.t[0]);

int num_iterations = defaultclock.i_end;
int current_iteration = defaultclock.timestep[0];
static int start_offset = current_iteration - _numt;

    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
addresses_monitor__dynamic_array_statemonitor_2_V.clear();
for(int i = 0; i < _num__array_statemonitor_2__indices; i++)
{
    _dynamic_array_statemonitor_2_V[i].resize(_numt + num_iterations - current_iteration);
    addresses_monitor__dynamic_array_statemonitor_2_V.push_back(thrust::raw_pointer_cast(&_dynamic_array_statemonitor_2_V[i][0]));
}
// Print a warning when the monitor is not going to work (#50)
if (_num__array_statemonitor_2__indices > 1024)
{
    printf("ERROR in statemonitor_2: Too many neurons recorded. Due to a bug (brian-team/brian2cuda#50), "
            "currently only as many neurons can be recorded as threads can be called from a single block!\n");
}



        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_statemonitor_2_codeobject))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_statemonitor_2_codeobject "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

        }
        first_run = false;
    }


// If the StateMonitor is run outside the MagicNetwork, we need to resize it.
// Happens e.g. when StateMonitor.record_single_timestep() is called.
if(current_iteration >= num_iterations)
{
    for(int i = 0; i < _num__array_statemonitor_2__indices; i++)
    {
        _dynamic_array_statemonitor_2_V[i].resize(_numt + 1);
        addresses_monitor__dynamic_array_statemonitor_2_V[i] = thrust::raw_pointer_cast(&_dynamic_array_statemonitor_2_V[i][0]);
    }
}

if (_num__array_statemonitor_2__indices > 0)
// TODO we get invalid launch configuration if this is 0, which happens e.g. for StateMonitor(..., variables=[])
{
    kernel_statemonitor_2_codeobject<<<1, _num__array_statemonitor_2__indices>>>(
        _num__array_statemonitor_2__indices,
        dev_array_statemonitor_2__indices,
        current_iteration - start_offset,
        thrust::raw_pointer_cast(&addresses_monitor__dynamic_array_statemonitor_2_V[0]),
        ///// HOST_PARAMETERS /////
        dev_array_neurongroup_3_V,
			_array_statemonitor_2_t,
			_numt,
			dev_array_statemonitor_2_N
        );

    CUDA_CHECK_ERROR("kernel_statemonitor_2_codeobject");
}


}


