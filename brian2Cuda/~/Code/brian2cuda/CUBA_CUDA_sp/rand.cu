#include "hip/hip_runtime.h"

#include "objects.h"
#include "rand.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/cuda_utils.h"
#include "network.h"
#include <hiprand.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

// XXX: for some documentation on random number generation, check out our wiki:
//      https://github.com/brian-team/brian2cuda/wiki/Random-number-generation

using namespace brian;

// TODO make this a class member function
// TODO don't call one kernel per codeobject but instead on kernel which takes
//      care of all codeobjects, preferably called with as many threads/blocks
//      as necessary for all states and initializing in parallel with warp
//      level divergence [needs changing set_curand_device_api_states()]
namespace {

    __global__ void init_curand_states(int N, int sequence_offset)
    {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N)
        {
            // Each thread gets the same seed, a different sequence number and
            // no offset
            // TODO: different seed and 0 sequence number is much faster, with
            // less security for independent sequences, add option as
            // preference!
            //hiprand_init(curand_seed + idx, 0, 0,
            hiprand_init(
                    *d_curand_seed,          // seed
                    sequence_offset + idx,   // sequence number
                    0,                       // offset
                    &d_curand_states[idx]);
        }
    }
}


// need a function pointer for Network::add(), can't pass a pointer to a class
// method, which is of different type
void _run_random_number_buffer()
{
    // random_number_buffer is a RandomNumberBuffer instance, declared in objects.cu
    random_number_buffer.next_time_step();
}


void RandomNumberBuffer::init()
{
    // check that we have enough memory available
    size_t free_byte;
    size_t total_byte;
    CUDA_SAFE_CALL(
            hipMemGetInfo(&free_byte, &total_byte)
            );
    size_t num_free_floats = free_byte / sizeof(randomNumber_t);

    if (run_counter == 0)
    {

        // number of time steps each codeobject is executed during current Network::run() call
        // XXX: we are assuming here that this function is only run in the first time step of a Network::run()
        int64_t num_steps_this_run_poissongroup_1_thresholder_codeobject = defaultclock.i_end - *(defaultclock.timestep);
        int64_t num_steps_this_run_poissongroup_thresholder_codeobject = defaultclock.i_end - *(defaultclock.timestep);

        // Get the number of needed random numbers per clock cycle, the generation interval, and the number generated per hiprand call.
        num_per_cycle_rand_poissongroup_1_thresholder_codeobject = 10 * 1;
        rand_floats_per_obj_poissongroup_1_thresholder_codeobject = floats_per_obj;
        if (floats_per_obj < num_per_cycle_rand_poissongroup_1_thresholder_codeobject)
            rand_floats_per_obj_poissongroup_1_thresholder_codeobject = num_per_cycle_rand_poissongroup_1_thresholder_codeobject;
        rand_interval_poissongroup_1_thresholder_codeobject = (int)(rand_floats_per_obj_poissongroup_1_thresholder_codeobject / num_per_cycle_rand_poissongroup_1_thresholder_codeobject);
        num_per_gen_rand_poissongroup_1_thresholder_codeobject = num_per_cycle_rand_poissongroup_1_thresholder_codeobject * rand_interval_poissongroup_1_thresholder_codeobject;
        idx_rand_poissongroup_1_thresholder_codeobject = rand_interval_poissongroup_1_thresholder_codeobject;

        // create max as many random numbers as will be needed during the current Network.run() call
        if ((int64_t)rand_interval_poissongroup_1_thresholder_codeobject > num_steps_this_run_poissongroup_1_thresholder_codeobject)
        {
            // NOTE: if the conditional is true, we can savely cast num_steps_this_run_poissongroup_1_thresholder_codeobject to int
            num_per_gen_rand_poissongroup_1_thresholder_codeobject = num_per_cycle_rand_poissongroup_1_thresholder_codeobject * (int)num_steps_this_run_poissongroup_1_thresholder_codeobject;
            assert((int64_t)num_per_cycle_rand_poissongroup_1_thresholder_codeobject * num_steps_this_run_poissongroup_1_thresholder_codeobject == num_per_gen_rand_poissongroup_1_thresholder_codeobject);
            rand_interval_poissongroup_1_thresholder_codeobject = (int)num_steps_this_run_poissongroup_1_thresholder_codeobject;
            // set this for buffer to be refilled at first next_time_step() call
            idx_rand_poissongroup_1_thresholder_codeobject = rand_interval_poissongroup_1_thresholder_codeobject;
        }

        // hiprandGenerateNormal requires an even number for pseudorandom generators
        if (num_per_gen_rand_poissongroup_1_thresholder_codeobject % 2 != 0)
        {
            num_per_gen_rand_poissongroup_1_thresholder_codeobject = num_per_gen_rand_poissongroup_1_thresholder_codeobject + 1;
        }

        // make sure that we don't use more memory then available
        // this checks per codeobject the number of generated floats against total available floats
        while (num_free_floats < num_per_gen_rand_poissongroup_1_thresholder_codeobject)
        {
            printf("INFO not enough memory available to generate %i random numbers for poissongroup_1_thresholder_codeobject, reducing the buffer size\n", num_free_floats);
            if (num_per_gen_rand_poissongroup_1_thresholder_codeobject < num_per_cycle_rand_poissongroup_1_thresholder_codeobject)
            {
                if (num_free_floats < num_per_cycle_rand_poissongroup_1_thresholder_codeobject)
                {
                    printf("ERROR not enough memory to generate random numbers for poissongroup_1_thresholder_codeobject %s:%d\n", __FILE__, __LINE__);
                    _dealloc_arrays();
                    exit(1);
                }
                else
                {
                    num_per_gen_rand_poissongroup_1_thresholder_codeobject = num_per_cycle_rand_poissongroup_1_thresholder_codeobject;
                    break;
                }
            }
            num_per_gen_rand_poissongroup_1_thresholder_codeobject /= 2;
        }
        printf("INFO generating %i rand every %i clock cycles for poissongroup_1_thresholder_codeobject\n", num_per_gen_rand_poissongroup_1_thresholder_codeobject, rand_interval_poissongroup_1_thresholder_codeobject);

        CUDA_SAFE_CALL(
                hipMalloc((void**)&dev_poissongroup_1_thresholder_codeobject_rand_allocator, sizeof(randomNumber_t)*num_per_gen_rand_poissongroup_1_thresholder_codeobject)
                );
        // Get the number of needed random numbers per clock cycle, the generation interval, and the number generated per hiprand call.
        num_per_cycle_rand_poissongroup_thresholder_codeobject = 100 * 1;
        rand_floats_per_obj_poissongroup_thresholder_codeobject = floats_per_obj;
        if (floats_per_obj < num_per_cycle_rand_poissongroup_thresholder_codeobject)
            rand_floats_per_obj_poissongroup_thresholder_codeobject = num_per_cycle_rand_poissongroup_thresholder_codeobject;
        rand_interval_poissongroup_thresholder_codeobject = (int)(rand_floats_per_obj_poissongroup_thresholder_codeobject / num_per_cycle_rand_poissongroup_thresholder_codeobject);
        num_per_gen_rand_poissongroup_thresholder_codeobject = num_per_cycle_rand_poissongroup_thresholder_codeobject * rand_interval_poissongroup_thresholder_codeobject;
        idx_rand_poissongroup_thresholder_codeobject = rand_interval_poissongroup_thresholder_codeobject;

        // create max as many random numbers as will be needed during the current Network.run() call
        if ((int64_t)rand_interval_poissongroup_thresholder_codeobject > num_steps_this_run_poissongroup_thresholder_codeobject)
        {
            // NOTE: if the conditional is true, we can savely cast num_steps_this_run_poissongroup_thresholder_codeobject to int
            num_per_gen_rand_poissongroup_thresholder_codeobject = num_per_cycle_rand_poissongroup_thresholder_codeobject * (int)num_steps_this_run_poissongroup_thresholder_codeobject;
            assert((int64_t)num_per_cycle_rand_poissongroup_thresholder_codeobject * num_steps_this_run_poissongroup_thresholder_codeobject == num_per_gen_rand_poissongroup_thresholder_codeobject);
            rand_interval_poissongroup_thresholder_codeobject = (int)num_steps_this_run_poissongroup_thresholder_codeobject;
            // set this for buffer to be refilled at first next_time_step() call
            idx_rand_poissongroup_thresholder_codeobject = rand_interval_poissongroup_thresholder_codeobject;
        }

        // hiprandGenerateNormal requires an even number for pseudorandom generators
        if (num_per_gen_rand_poissongroup_thresholder_codeobject % 2 != 0)
        {
            num_per_gen_rand_poissongroup_thresholder_codeobject = num_per_gen_rand_poissongroup_thresholder_codeobject + 1;
        }

        // make sure that we don't use more memory then available
        // this checks per codeobject the number of generated floats against total available floats
        while (num_free_floats < num_per_gen_rand_poissongroup_thresholder_codeobject)
        {
            printf("INFO not enough memory available to generate %i random numbers for poissongroup_thresholder_codeobject, reducing the buffer size\n", num_free_floats);
            if (num_per_gen_rand_poissongroup_thresholder_codeobject < num_per_cycle_rand_poissongroup_thresholder_codeobject)
            {
                if (num_free_floats < num_per_cycle_rand_poissongroup_thresholder_codeobject)
                {
                    printf("ERROR not enough memory to generate random numbers for poissongroup_thresholder_codeobject %s:%d\n", __FILE__, __LINE__);
                    _dealloc_arrays();
                    exit(1);
                }
                else
                {
                    num_per_gen_rand_poissongroup_thresholder_codeobject = num_per_cycle_rand_poissongroup_thresholder_codeobject;
                    break;
                }
            }
            num_per_gen_rand_poissongroup_thresholder_codeobject /= 2;
        }
        printf("INFO generating %i rand every %i clock cycles for poissongroup_thresholder_codeobject\n", num_per_gen_rand_poissongroup_thresholder_codeobject, rand_interval_poissongroup_thresholder_codeobject);

        CUDA_SAFE_CALL(
                hipMalloc((void**)&dev_poissongroup_thresholder_codeobject_rand_allocator, sizeof(randomNumber_t)*num_per_gen_rand_poissongroup_thresholder_codeobject)
                );



        // now check if the total number of generated floats fit into available memory
        int total_num_generated_floats = 0;
        total_num_generated_floats += num_per_gen_rand_poissongroup_1_thresholder_codeobject;
        total_num_generated_floats += num_per_gen_rand_poissongroup_thresholder_codeobject;
        if (num_free_floats < total_num_generated_floats)
        {
            // TODO: find a way to deal with this? E.g. looping over buffers sorted
            // by buffer size and reducing them until it fits.
            printf("MEMORY ERROR: Trying to generate more random numbers than fit "
                   "into available memory. Please report this as an issue on "
                   "GitHub: https://github.com/brian-team/brian2cuda/issues/new");
            _dealloc_arrays();
            exit(1);
        }

    } // if (run_counter == 0)

    // init hiprand states only in first run
    if (run_counter == 0)
    {

        // Update hiprand device api states once before anything is run. At this
        // point all N's (also from probabilistically generated synapses) are
        // known. This might update the number of needed hiprand states.
        ensure_enough_curand_states();
    }

}


void RandomNumberBuffer::allocate_device_curand_states()
{
    // allocate globabl memory for hiprand device api states
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_curand_states,
                sizeof(hiprandState) * num_curand_states)
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_curand_states),
                &dev_curand_states, sizeof(hiprandState*))
            );
}



void RandomNumberBuffer::update_needed_number_curand_states()
{
    // Find the maximum number of threads generating random numbers in parallel
    // using the cuRAND device API. For synapses objects, the number of
    // synapses might not be known yet. This is the case when the first random
    // seed is set and for any seed() call before the synapses creation.
    num_threads_curand_init = max_threads_per_block;
    num_blocks_curand_init = num_curand_states / max_threads_per_block + 1;
    if (num_curand_states < num_threads_curand_init)
        num_threads_curand_init = num_curand_states;
}


void RandomNumberBuffer::set_curand_device_api_states(bool reset_seed)
{
    int sequence_offset = 0;
    int num_curand_states_old = num_curand_states;
    // Whenever hiprand states are set, check if enough states where
    // initialized. This will generate states the first time the seed is set.
    // But it can be that the seed is set before all network objects' N are
    // available (e.g. synapses not created yet) and before the network is
    // run. In such a case, once the network is run, missing hiprand states are
    // generated here. If the seed was not reset inbetween, the pervious states
    // should not be reinitialized (achieved by the `sequence_offset`
    // parameter). If the seed was reset, then all states should be
    // reinitialized.
    update_needed_number_curand_states();

    // number of hiprand states that need to be initialized
    int num_curand_states_to_init;

    if (reset_seed)
    {
        // initialize all hiprand states
        num_curand_states_to_init = num_curand_states;
        sequence_offset = 0;
    }
    else
    {
        // don't initialize existing hiprand states, only the new ones
        num_curand_states_to_init = num_curand_states - num_curand_states_old;
        sequence_offset = num_curand_states_old;
    }

    if (num_curand_states_old < num_curand_states)
    {
        // copy hiprand states to new array of updated size
        hiprandState* dev_curand_states_old = dev_curand_states;
        // allocate memory for new number of hiprand states
        allocate_device_curand_states();

        if ((!reset_seed) && (num_curand_states_old > 0))
        {
            // copy old states to new memory address on device
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_curand_states, dev_curand_states_old,
                        sizeof(hiprandState) * num_curand_states_old,
                        hipMemcpyDeviceToDevice)
                    );
        }
    }

    if (num_curand_states_to_init > 0)
    {
        init_curand_states<<<num_blocks_curand_init, num_threads_curand_init>>>(
                num_curand_states_to_init,
                sequence_offset);
    }
}


void RandomNumberBuffer::ensure_enough_curand_states()
{
    // Separate public function needed for synapses codeobjects that are run
    // only once before the network
    // The N of synapses will not be known when setting the seed and needs to
    // be updated before using random numbers per synapse. This occurs e.g.
    // when initializing synaptic variables (synapses_group_conditional_....)
    bool reset_seed = false;
    set_curand_device_api_states(reset_seed);
}


void RandomNumberBuffer::run_finished()
{
    needs_init = true;
    run_counter += 1;
}


void RandomNumberBuffer::set_seed(unsigned long long seed)
{
    CUDA_SAFE_CALL(
            hiprandSetPseudoRandomGeneratorSeed(curand_generator, seed)
            );

    // generator offset needs to be reset to its default (=0)
    CUDA_SAFE_CALL(
            hiprandSetGeneratorOffset(curand_generator, 0ULL)
            );

    // set seed for hiprand device api calls
    // don't set the same seed for host api and device api random states, just in case
    unsigned long long curand_seed = seed + 1;
    CUDA_SAFE_CALL(
            hipMemcpy(dev_curand_seed, &curand_seed,
                sizeof(unsigned long long), hipMemcpyHostToDevice)
            );

    bool reset_seed = true;
    set_curand_device_api_states(reset_seed);
    // We set all device api states for codeobjects run outside the network
    // since we don't know when they will be used.
    //set_curand_device_api_states_for_separate_calls();
    // Curand device api states for binomials during network runs will be set
    // only for the current run in init(), once the network starts.
}


void RandomNumberBuffer::refill_uniform_numbers(
        randomNumber_t* dev_rand_allocator,
        randomNumber_t* &dev_rand,
        int num_per_gen_rand,
        int &idx_rand)
{
    // generate uniform distributed random numbers and reset buffer index

    hiprandGenerateUniformDouble(curand_generator, dev_rand_allocator, num_per_gen_rand);
    // before: XXX dev_rand = &dev_rand_allocator[0];
    dev_rand = dev_rand_allocator;
    idx_rand = 1;
}


void RandomNumberBuffer::refill_normal_numbers(
        randomNumber_t* dev_randn_allocator,
        randomNumber_t* &dev_randn,
        int num_per_gen_randn,
        int &idx_randn)
{
    // generate normal distributed random numbers and reset buffer index

    hiprandGenerateNormalDouble(curand_generator, dev_randn_allocator, num_per_gen_randn, 0, 1);
    // before: XXX dev_randn = &dev_randn_allocator[0];
    dev_randn = dev_randn_allocator;
    idx_randn = 1;
}


void RandomNumberBuffer::next_time_step()
{
    // init buffers at fist time step of each run call
    if (needs_init)
    {
        // free device memory for random numbers used during last run call
        if (run_counter > 0)
        {
            if (run_counter == 0)
            {

                CUDA_SAFE_CALL(
                        hipFree(dev_poissongroup_1_thresholder_codeobject_rand_allocator)
                        );
                CUDA_SAFE_CALL(
                        hipFree(dev_poissongroup_thresholder_codeobject_rand_allocator)
                        );

            } // run_counter == 0
        }

        // init random number buffers
        init();
        needs_init = false;
    }

    if (run_counter == 0)
    {

        // uniform numbers for poissongroup_1_thresholder_codeobject
        if (idx_rand_poissongroup_1_thresholder_codeobject == rand_interval_poissongroup_1_thresholder_codeobject)
        {
            refill_uniform_numbers(
                    dev_poissongroup_1_thresholder_codeobject_rand_allocator,
                    dev_poissongroup_1_thresholder_codeobject_rand,
                    num_per_gen_rand_poissongroup_1_thresholder_codeobject,
                    idx_rand_poissongroup_1_thresholder_codeobject);
        }
        else
        {
            // move device pointer to next numbers
            dev_poissongroup_1_thresholder_codeobject_rand += num_per_cycle_rand_poissongroup_1_thresholder_codeobject;
            idx_rand_poissongroup_1_thresholder_codeobject += 1;
        }
        assert(dev_poissongroup_1_thresholder_codeobject_rand < dev_poissongroup_1_thresholder_codeobject_rand_allocator + num_per_gen_rand_poissongroup_1_thresholder_codeobject);
        // uniform numbers for poissongroup_thresholder_codeobject
        if (idx_rand_poissongroup_thresholder_codeobject == rand_interval_poissongroup_thresholder_codeobject)
        {
            refill_uniform_numbers(
                    dev_poissongroup_thresholder_codeobject_rand_allocator,
                    dev_poissongroup_thresholder_codeobject_rand,
                    num_per_gen_rand_poissongroup_thresholder_codeobject,
                    idx_rand_poissongroup_thresholder_codeobject);
        }
        else
        {
            // move device pointer to next numbers
            dev_poissongroup_thresholder_codeobject_rand += num_per_cycle_rand_poissongroup_thresholder_codeobject;
            idx_rand_poissongroup_thresholder_codeobject += 1;
        }
        assert(dev_poissongroup_thresholder_codeobject_rand < dev_poissongroup_thresholder_codeobject_rand_allocator + num_per_gen_rand_poissongroup_thresholder_codeobject);

    }// run_counter == 0
}
