#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "objects.h"
#include <ctime>
#include <time.h>
#include "run.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "rand.h"

#include "code_objects/synapses_pre_push_spikes.h"
#include "code_objects/synapses_5_post_codeobject.h"
#include "code_objects/synapses_2_post_initialise_queue.h"
#include "code_objects/synapses_pre_codeobject.h"
#include "code_objects/neurongroup_2_resetter_codeobject.h"
#include "code_objects/neurongroup_2_stateupdater_codeobject.h"
#include "code_objects/synapses_4_pre_initialise_queue.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_6.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_7.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_4.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_5.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_2.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_3.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_1.h"
#include "code_objects/synapses_1_post_push_spikes.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_8.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject_9.h"
#include "code_objects/synapses_4_synapses_create_array_codeobject.h"
#include "code_objects/synapses_post_initialise_queue.h"
#include "code_objects/spikemonitor_3_codeobject.h"
#include "code_objects/synapses_2_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_5_pre_codeobject.h"
#include "code_objects/synapses_1_pre_push_spikes.h"
#include "code_objects/neurongroup_3_resetter_codeobject.h"
#include "code_objects/synapses_2_post_push_spikes.h"
#include "code_objects/neurongroup_1_stateupdater_codeobject.h"
#include "code_objects/ratemonitor_1_codeobject.h"
#include "code_objects/neurongroup_2_thresholder_codeobject.h"
#include "code_objects/synapses_3_pre_codeobject.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_3.h"
#include "code_objects/synapses_1_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_4_pre_codeobject.h"
#include "code_objects/synapses_2_post_codeobject.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject.h"
#include "code_objects/synapses_post_codeobject.h"
#include "code_objects/neurongroup_1_resetter_codeobject.h"
#include "code_objects/synapses_1_pre_initialise_queue.h"
#include "code_objects/ratemonitor_3_codeobject.h"
#include "code_objects/synapses_1_post_initialise_queue.h"
#include "code_objects/synapses_6_pre_codeobject.h"
#include "code_objects/statemonitor_2_codeobject.h"
#include "code_objects/poissongroup_thresholder_codeobject.h"
#include "code_objects/synapses_pre_initialise_queue.h"
#include "code_objects/neurongroup_resetter_codeobject.h"
#include "code_objects/synapses_4_post_initialise_queue.h"
#include "code_objects/synapses_5_post_initialise_queue.h"
#include "code_objects/statemonitor_3_codeobject.h"
#include "code_objects/synapses_5_pre_initialise_queue.h"
#include "code_objects/statemonitor_1_codeobject.h"
#include "code_objects/ratemonitor_codeobject.h"
#include "code_objects/neurongroup_stateupdater_codeobject.h"
#include "code_objects/ratemonitor_2_codeobject.h"
#include "code_objects/spikemonitor_1_codeobject.h"
#include "code_objects/poissongroup_1_thresholder_codeobject.h"
#include "code_objects/neurongroup_thresholder_codeobject.h"
#include "code_objects/synapses_6_pre_initialise_queue.h"
#include "code_objects/synapses_4_pre_push_spikes.h"
#include "code_objects/synapses_6_pre_push_spikes.h"
#include "code_objects/neurongroup_3_thresholder_codeobject.h"
#include "code_objects/synapses_3_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_post_push_spikes.h"
#include "code_objects/synapses_5_post_push_spikes.h"
#include "code_objects/synapses_2_pre_initialise_queue.h"
#include "code_objects/synapses_3_pre_initialise_queue.h"
#include "code_objects/synapses_6_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_4_post_codeobject.h"
#include "code_objects/neurongroup_1_thresholder_codeobject.h"
#include "code_objects/synapses_2_pre_codeobject.h"
#include "code_objects/synapses_2_pre_push_spikes.h"
#include "code_objects/statemonitor_codeobject.h"
#include "code_objects/synapses_3_pre_push_spikes.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_5.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_4.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_7.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_6.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_1.h"
#include "code_objects/spikemonitor_codeobject.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_2.h"
#include "code_objects/synapses_5_pre_push_spikes.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_9.h"
#include "code_objects/synapses_5_synapses_create_array_codeobject_8.h"
#include "code_objects/synapses_1_post_codeobject.h"
#include "code_objects/synapses_4_post_push_spikes.h"
#include "code_objects/synapses_synapses_create_generator_codeobject.h"
#include "code_objects/neurongroup_3_stateupdater_codeobject.h"
#include "code_objects/spikemonitor_2_codeobject.h"
#include "code_objects/synapses_1_pre_codeobject.h"


#include <iostream>
#include <fstream>
#include "hip/hip_runtime_api.h"




int main(int argc, char **argv)
{
    // seed variable set in Python through brian2.seed() calls can use this
    // variable (see device.py CUDAStandaloneDevice.generate_main_source())
    unsigned long long seed;

    const std::clock_t _start_time = std::clock();

    const std::clock_t _start_time2 = std::clock();

    hipDeviceProp_t props;
    CUDA_SAFE_CALL(
            hipGetDeviceProperties(&props, 0)
            );
    size_t limit = 128 * 1024 * 1024;
    CUDA_SAFE_CALL(
            hipDeviceSetLimit(hipLimitMallocHeapSize, limit)
            );
    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );

    const double _run_time2 = (double)(std::clock() -_start_time2)/CLOCKS_PER_SEC;
    printf("INFO: setting cudaDevice stuff took %f seconds\n", _run_time2);

    brian_start();

    const std::clock_t _start_time3 = std::clock();
    {
        using namespace brian;

                
                        for(int i=0; i<_num__array_neurongroup_1__spikespace; i++)
                        {
                            _array_neurongroup_1__spikespace[i] = -1;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup_1__spikespace[current_idx_array_neurongroup_1__spikespace], &_array_neurongroup_1__spikespace[0],
                                        sizeof(_array_neurongroup_1__spikespace[0])*_num__array_neurongroup_1__spikespace, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_neurongroup_3__spikespace; i++)
                        {
                            _array_neurongroup_3__spikespace[i] = -1;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup_3__spikespace[current_idx_array_neurongroup_3__spikespace], &_array_neurongroup_3__spikespace[0],
                                        sizeof(_array_neurongroup_3__spikespace[0])*_num__array_neurongroup_3__spikespace, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_neurongroup__spikespace; i++)
                        {
                            _array_neurongroup__spikespace[i] = -1;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup__spikespace[current_idx_array_neurongroup__spikespace], &_array_neurongroup__spikespace[0],
                                        sizeof(_array_neurongroup__spikespace[0])*_num__array_neurongroup__spikespace, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_poissongroup_1__spikespace; i++)
                        {
                            _array_poissongroup_1__spikespace[i] = -1;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_poissongroup_1__spikespace[current_idx_array_poissongroup_1__spikespace], &_array_poissongroup_1__spikespace[0],
                                        sizeof(_array_poissongroup_1__spikespace[0])*_num__array_poissongroup_1__spikespace, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_poissongroup__spikespace; i++)
                        {
                            _array_poissongroup__spikespace[i] = -1;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_poissongroup__spikespace[current_idx_array_poissongroup__spikespace], &_array_poissongroup__spikespace[0],
                                        sizeof(_array_poissongroup__spikespace[0])*_num__array_poissongroup__spikespace, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_neurongroup_2__spikespace; i++)
                        {
                            _array_neurongroup_2__spikespace[i] = -1;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup_2__spikespace[current_idx_array_neurongroup_2__spikespace], &_array_neurongroup_2__spikespace[0],
                                        sizeof(_array_neurongroup_2__spikespace[0])*_num__array_neurongroup_2__spikespace, hipMemcpyHostToDevice)
                                );
                        
        
                        _array_defaultclock_dt[0] = 0.0001;
                        CUDA_SAFE_CALL(
                                hipMemcpy(&dev_array_defaultclock_dt[0], &_array_defaultclock_dt[0],
                                        sizeof(_array_defaultclock_dt[0]), hipMemcpyHostToDevice)
                                );
                        
        
                        _array_defaultclock_dt[0] = 0.0001;
                        CUDA_SAFE_CALL(
                                hipMemcpy(&dev_array_defaultclock_dt[0], &_array_defaultclock_dt[0],
                                        sizeof(_array_defaultclock_dt[0]), hipMemcpyHostToDevice)
                                );
                        
        
                        _array_defaultclock_dt[0] = 0.0001;
                        CUDA_SAFE_CALL(
                                hipMemcpy(&dev_array_defaultclock_dt[0], &_array_defaultclock_dt[0],
                                        sizeof(_array_defaultclock_dt[0]), hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_neurongroup_V; i++)
                        {
                            _array_neurongroup_V[i] = -0.058;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup_V, &_array_neurongroup_V[0],
                                        sizeof(_array_neurongroup_V[0])*_num__array_neurongroup_V, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_synapses_create_generator_codeobject();
        
                        for(int i=0; i<_dynamic_array_synapses_w_MFGR.size(); i++)
                        {
                            _dynamic_array_synapses_w_MFGR[i] = 4;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(thrust::raw_pointer_cast(&dev_dynamic_array_synapses_w_MFGR[0]), &_dynamic_array_synapses_w_MFGR[0],
                                        sizeof(_dynamic_array_synapses_w_MFGR[0])*_dynamic_array_synapses_w_MFGR.size(), hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_neurongroup_1_V; i++)
                        {
                            _array_neurongroup_1_V[i] = -0.055;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup_1_V, &_array_neurongroup_1_V[0],
                                        sizeof(_array_neurongroup_1_V[0])*_num__array_neurongroup_1_V, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_neurongroup_2_V; i++)
                        {
                            _array_neurongroup_2_V[i] = -0.068;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup_2_V, &_array_neurongroup_2_V[0],
                                        sizeof(_array_neurongroup_2_V[0])*_num__array_neurongroup_2_V, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_1_synapses_create_generator_codeobject();
        
                        for(int i=0; i<_dynamic_array_synapses_1_w_CFPKJ.size(); i++)
                        {
                            _dynamic_array_synapses_1_w_CFPKJ[i] = 1.0;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1_w_CFPKJ[0]), &_dynamic_array_synapses_1_w_CFPKJ[0],
                                        sizeof(_dynamic_array_synapses_1_w_CFPKJ[0])*_dynamic_array_synapses_1_w_CFPKJ.size(), hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__array_neurongroup_3_V; i++)
                        {
                            _array_neurongroup_3_V[i] = -0.068;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_neurongroup_3_V, &_array_neurongroup_3_V[0],
                                        sizeof(_array_neurongroup_3_V[0])*_num__array_neurongroup_3_V, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_2_synapses_create_generator_codeobject();
        
                        for(int i=0; i<_dynamic_array_synapses_2_w_GRGO.size(); i++)
                        {
                            _dynamic_array_synapses_2_w_GRGO[i] = 4e-05;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(thrust::raw_pointer_cast(&dev_dynamic_array_synapses_2_w_GRGO[0]), &_dynamic_array_synapses_2_w_GRGO[0],
                                        sizeof(_dynamic_array_synapses_2_w_GRGO[0])*_dynamic_array_synapses_2_w_GRGO.size(), hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_3_synapses_create_generator_codeobject();
        
                        for(int i=0; i<_dynamic_array_synapses_3_w_GOGR.size(); i++)
                        {
                            _dynamic_array_synapses_3_w_GOGR[i] = 10;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(thrust::raw_pointer_cast(&dev_dynamic_array_synapses_3_w_GOGR[0]), &_dynamic_array_synapses_3_w_GOGR[0],
                                        sizeof(_dynamic_array_synapses_3_w_GOGR[0])*_dynamic_array_synapses_3_w_GOGR.size(), hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources; i++)
                        {
                            _array_synapses_4_sources[i] = _static_array__array_synapses_4_sources[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources, &_array_synapses_4_sources[0],
                                        sizeof(_array_synapses_4_sources[0])*_num__array_synapses_4_sources, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets; i++)
                        {
                            _array_synapses_4_targets[i] = _static_array__array_synapses_4_targets[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets, &_array_synapses_4_targets[0],
                                        sizeof(_array_synapses_4_targets[0])*_num__array_synapses_4_targets, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_1; i++)
                        {
                            _array_synapses_4_sources_1[i] = _static_array__array_synapses_4_sources_1[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_1, &_array_synapses_4_sources_1[0],
                                        sizeof(_array_synapses_4_sources_1[0])*_num__array_synapses_4_sources_1, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_1; i++)
                        {
                            _array_synapses_4_targets_1[i] = _static_array__array_synapses_4_targets_1[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_1, &_array_synapses_4_targets_1[0],
                                        sizeof(_array_synapses_4_targets_1[0])*_num__array_synapses_4_targets_1, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_1();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_2; i++)
                        {
                            _array_synapses_4_sources_2[i] = _static_array__array_synapses_4_sources_2[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_2, &_array_synapses_4_sources_2[0],
                                        sizeof(_array_synapses_4_sources_2[0])*_num__array_synapses_4_sources_2, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_2; i++)
                        {
                            _array_synapses_4_targets_2[i] = _static_array__array_synapses_4_targets_2[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_2, &_array_synapses_4_targets_2[0],
                                        sizeof(_array_synapses_4_targets_2[0])*_num__array_synapses_4_targets_2, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_2();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_3; i++)
                        {
                            _array_synapses_4_sources_3[i] = _static_array__array_synapses_4_sources_3[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_3, &_array_synapses_4_sources_3[0],
                                        sizeof(_array_synapses_4_sources_3[0])*_num__array_synapses_4_sources_3, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_3; i++)
                        {
                            _array_synapses_4_targets_3[i] = _static_array__array_synapses_4_targets_3[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_3, &_array_synapses_4_targets_3[0],
                                        sizeof(_array_synapses_4_targets_3[0])*_num__array_synapses_4_targets_3, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_3();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_4; i++)
                        {
                            _array_synapses_4_sources_4[i] = _static_array__array_synapses_4_sources_4[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_4, &_array_synapses_4_sources_4[0],
                                        sizeof(_array_synapses_4_sources_4[0])*_num__array_synapses_4_sources_4, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_4; i++)
                        {
                            _array_synapses_4_targets_4[i] = _static_array__array_synapses_4_targets_4[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_4, &_array_synapses_4_targets_4[0],
                                        sizeof(_array_synapses_4_targets_4[0])*_num__array_synapses_4_targets_4, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_4();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_5; i++)
                        {
                            _array_synapses_4_sources_5[i] = _static_array__array_synapses_4_sources_5[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_5, &_array_synapses_4_sources_5[0],
                                        sizeof(_array_synapses_4_sources_5[0])*_num__array_synapses_4_sources_5, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_5; i++)
                        {
                            _array_synapses_4_targets_5[i] = _static_array__array_synapses_4_targets_5[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_5, &_array_synapses_4_targets_5[0],
                                        sizeof(_array_synapses_4_targets_5[0])*_num__array_synapses_4_targets_5, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_5();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_6; i++)
                        {
                            _array_synapses_4_sources_6[i] = _static_array__array_synapses_4_sources_6[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_6, &_array_synapses_4_sources_6[0],
                                        sizeof(_array_synapses_4_sources_6[0])*_num__array_synapses_4_sources_6, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_6; i++)
                        {
                            _array_synapses_4_targets_6[i] = _static_array__array_synapses_4_targets_6[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_6, &_array_synapses_4_targets_6[0],
                                        sizeof(_array_synapses_4_targets_6[0])*_num__array_synapses_4_targets_6, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_6();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_7; i++)
                        {
                            _array_synapses_4_sources_7[i] = _static_array__array_synapses_4_sources_7[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_7, &_array_synapses_4_sources_7[0],
                                        sizeof(_array_synapses_4_sources_7[0])*_num__array_synapses_4_sources_7, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_7; i++)
                        {
                            _array_synapses_4_targets_7[i] = _static_array__array_synapses_4_targets_7[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_7, &_array_synapses_4_targets_7[0],
                                        sizeof(_array_synapses_4_targets_7[0])*_num__array_synapses_4_targets_7, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_7();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_8; i++)
                        {
                            _array_synapses_4_sources_8[i] = _static_array__array_synapses_4_sources_8[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_8, &_array_synapses_4_sources_8[0],
                                        sizeof(_array_synapses_4_sources_8[0])*_num__array_synapses_4_sources_8, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_8; i++)
                        {
                            _array_synapses_4_targets_8[i] = _static_array__array_synapses_4_targets_8[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_8, &_array_synapses_4_targets_8[0],
                                        sizeof(_array_synapses_4_targets_8[0])*_num__array_synapses_4_targets_8, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_8();
        
                        for(int i=0; i<_num__static_array__array_synapses_4_sources_9; i++)
                        {
                            _array_synapses_4_sources_9[i] = _static_array__array_synapses_4_sources_9[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_sources_9, &_array_synapses_4_sources_9[0],
                                        sizeof(_array_synapses_4_sources_9[0])*_num__array_synapses_4_sources_9, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_4_targets_9; i++)
                        {
                            _array_synapses_4_targets_9[i] = _static_array__array_synapses_4_targets_9[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_4_targets_9, &_array_synapses_4_targets_9[0],
                                        sizeof(_array_synapses_4_targets_9[0])*_num__array_synapses_4_targets_9, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_4_synapses_create_array_codeobject_9();
        
                        for(int i=0; i<_dynamic_array_synapses_4_w_GRPKJ.size(); i++)
                        {
                            _dynamic_array_synapses_4_w_GRPKJ[i] = 0.003;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(thrust::raw_pointer_cast(&dev_dynamic_array_synapses_4_w_GRPKJ[0]), &_dynamic_array_synapses_4_w_GRPKJ[0],
                                        sizeof(_dynamic_array_synapses_4_w_GRPKJ[0])*_dynamic_array_synapses_4_w_GRPKJ.size(), hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources; i++)
                        {
                            _array_synapses_5_sources[i] = _static_array__array_synapses_5_sources[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources, &_array_synapses_5_sources[0],
                                        sizeof(_array_synapses_5_sources[0])*_num__array_synapses_5_sources, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets; i++)
                        {
                            _array_synapses_5_targets[i] = _static_array__array_synapses_5_targets[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets, &_array_synapses_5_targets[0],
                                        sizeof(_array_synapses_5_targets[0])*_num__array_synapses_5_targets, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_1; i++)
                        {
                            _array_synapses_5_sources_1[i] = _static_array__array_synapses_5_sources_1[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_1, &_array_synapses_5_sources_1[0],
                                        sizeof(_array_synapses_5_sources_1[0])*_num__array_synapses_5_sources_1, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_1; i++)
                        {
                            _array_synapses_5_targets_1[i] = _static_array__array_synapses_5_targets_1[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_1, &_array_synapses_5_targets_1[0],
                                        sizeof(_array_synapses_5_targets_1[0])*_num__array_synapses_5_targets_1, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_1();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_2; i++)
                        {
                            _array_synapses_5_sources_2[i] = _static_array__array_synapses_5_sources_2[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_2, &_array_synapses_5_sources_2[0],
                                        sizeof(_array_synapses_5_sources_2[0])*_num__array_synapses_5_sources_2, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_2; i++)
                        {
                            _array_synapses_5_targets_2[i] = _static_array__array_synapses_5_targets_2[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_2, &_array_synapses_5_targets_2[0],
                                        sizeof(_array_synapses_5_targets_2[0])*_num__array_synapses_5_targets_2, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_2();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_3; i++)
                        {
                            _array_synapses_5_sources_3[i] = _static_array__array_synapses_5_sources_3[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_3, &_array_synapses_5_sources_3[0],
                                        sizeof(_array_synapses_5_sources_3[0])*_num__array_synapses_5_sources_3, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_3; i++)
                        {
                            _array_synapses_5_targets_3[i] = _static_array__array_synapses_5_targets_3[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_3, &_array_synapses_5_targets_3[0],
                                        sizeof(_array_synapses_5_targets_3[0])*_num__array_synapses_5_targets_3, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_3();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_4; i++)
                        {
                            _array_synapses_5_sources_4[i] = _static_array__array_synapses_5_sources_4[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_4, &_array_synapses_5_sources_4[0],
                                        sizeof(_array_synapses_5_sources_4[0])*_num__array_synapses_5_sources_4, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_4; i++)
                        {
                            _array_synapses_5_targets_4[i] = _static_array__array_synapses_5_targets_4[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_4, &_array_synapses_5_targets_4[0],
                                        sizeof(_array_synapses_5_targets_4[0])*_num__array_synapses_5_targets_4, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_4();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_5; i++)
                        {
                            _array_synapses_5_sources_5[i] = _static_array__array_synapses_5_sources_5[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_5, &_array_synapses_5_sources_5[0],
                                        sizeof(_array_synapses_5_sources_5[0])*_num__array_synapses_5_sources_5, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_5; i++)
                        {
                            _array_synapses_5_targets_5[i] = _static_array__array_synapses_5_targets_5[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_5, &_array_synapses_5_targets_5[0],
                                        sizeof(_array_synapses_5_targets_5[0])*_num__array_synapses_5_targets_5, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_5();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_6; i++)
                        {
                            _array_synapses_5_sources_6[i] = _static_array__array_synapses_5_sources_6[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_6, &_array_synapses_5_sources_6[0],
                                        sizeof(_array_synapses_5_sources_6[0])*_num__array_synapses_5_sources_6, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_6; i++)
                        {
                            _array_synapses_5_targets_6[i] = _static_array__array_synapses_5_targets_6[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_6, &_array_synapses_5_targets_6[0],
                                        sizeof(_array_synapses_5_targets_6[0])*_num__array_synapses_5_targets_6, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_6();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_7; i++)
                        {
                            _array_synapses_5_sources_7[i] = _static_array__array_synapses_5_sources_7[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_7, &_array_synapses_5_sources_7[0],
                                        sizeof(_array_synapses_5_sources_7[0])*_num__array_synapses_5_sources_7, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_7; i++)
                        {
                            _array_synapses_5_targets_7[i] = _static_array__array_synapses_5_targets_7[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_7, &_array_synapses_5_targets_7[0],
                                        sizeof(_array_synapses_5_targets_7[0])*_num__array_synapses_5_targets_7, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_7();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_8; i++)
                        {
                            _array_synapses_5_sources_8[i] = _static_array__array_synapses_5_sources_8[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_8, &_array_synapses_5_sources_8[0],
                                        sizeof(_array_synapses_5_sources_8[0])*_num__array_synapses_5_sources_8, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_8; i++)
                        {
                            _array_synapses_5_targets_8[i] = _static_array__array_synapses_5_targets_8[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_8, &_array_synapses_5_targets_8[0],
                                        sizeof(_array_synapses_5_targets_8[0])*_num__array_synapses_5_targets_8, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_8();
        
                        for(int i=0; i<_num__static_array__array_synapses_5_sources_9; i++)
                        {
                            _array_synapses_5_sources_9[i] = _static_array__array_synapses_5_sources_9[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_sources_9, &_array_synapses_5_sources_9[0],
                                        sizeof(_array_synapses_5_sources_9[0])*_num__array_synapses_5_sources_9, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_synapses_5_targets_9; i++)
                        {
                            _array_synapses_5_targets_9[i] = _static_array__array_synapses_5_targets_9[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_synapses_5_targets_9, &_array_synapses_5_targets_9[0],
                                        sizeof(_array_synapses_5_targets_9[0])*_num__array_synapses_5_targets_9, hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_5_synapses_create_array_codeobject_9();
        
                        for(int i=0; i<_dynamic_array_synapses_5_w_GRBS.size(); i++)
                        {
                            _dynamic_array_synapses_5_w_GRBS[i] = 0.003;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(thrust::raw_pointer_cast(&dev_dynamic_array_synapses_5_w_GRBS[0]), &_dynamic_array_synapses_5_w_GRBS[0],
                                        sizeof(_dynamic_array_synapses_5_w_GRBS[0])*_dynamic_array_synapses_5_w_GRBS.size(), hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_6_synapses_create_generator_codeobject();
        
                        for(int i=0; i<_dynamic_array_synapses_6_w_BSPKJ.size(); i++)
                        {
                            _dynamic_array_synapses_6_w_BSPKJ[i] = 5.3;
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(thrust::raw_pointer_cast(&dev_dynamic_array_synapses_6_w_BSPKJ[0]), &_dynamic_array_synapses_6_w_BSPKJ[0],
                                        sizeof(_dynamic_array_synapses_6_w_BSPKJ[0])*_dynamic_array_synapses_6_w_BSPKJ.size(), hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_statemonitor__indices; i++)
                        {
                            _array_statemonitor__indices[i] = _static_array__array_statemonitor__indices[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_statemonitor__indices, &_array_statemonitor__indices[0],
                                        sizeof(_array_statemonitor__indices[0])*_num__array_statemonitor__indices, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_statemonitor_1__indices; i++)
                        {
                            _array_statemonitor_1__indices[i] = _static_array__array_statemonitor_1__indices[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_statemonitor_1__indices, &_array_statemonitor_1__indices[0],
                                        sizeof(_array_statemonitor_1__indices[0])*_num__array_statemonitor_1__indices, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_statemonitor_2__indices; i++)
                        {
                            _array_statemonitor_2__indices[i] = _static_array__array_statemonitor_2__indices[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_statemonitor_2__indices, &_array_statemonitor_2__indices[0],
                                        sizeof(_array_statemonitor_2__indices[0])*_num__array_statemonitor_2__indices, hipMemcpyHostToDevice)
                                );
                        
        
                        for(int i=0; i<_num__static_array__array_statemonitor_3__indices; i++)
                        {
                            _array_statemonitor_3__indices[i] = _static_array__array_statemonitor_3__indices[i];
                        }
                        
        
                        CUDA_SAFE_CALL(
                                hipMemcpy(dev_array_statemonitor_3__indices, &_array_statemonitor_3__indices[0],
                                        sizeof(_array_statemonitor_3__indices[0])*_num__array_statemonitor_3__indices, hipMemcpyHostToDevice)
                                );
                        
        
                        _array_defaultclock_timestep[0] = 0;
                        CUDA_SAFE_CALL(
                                hipMemcpy(&dev_array_defaultclock_timestep[0], &_array_defaultclock_timestep[0],
                                        sizeof(_array_defaultclock_timestep[0]), hipMemcpyHostToDevice)
                                );
                        
        
                        _array_defaultclock_t[0] = 0.0;
                        CUDA_SAFE_CALL(
                                hipMemcpy(&dev_array_defaultclock_t[0], &_array_defaultclock_t[0],
                                        sizeof(_array_defaultclock_t[0]), hipMemcpyHostToDevice)
                                );
                        
        _run_synapses_1_pre_initialise_queue();
        _run_synapses_2_pre_initialise_queue();
        _run_synapses_3_pre_initialise_queue();
        _run_synapses_4_pre_initialise_queue();
        _run_synapses_5_pre_initialise_queue();
        _run_synapses_6_pre_initialise_queue();
        _run_synapses_pre_initialise_queue();
        _run_synapses_1_post_initialise_queue();
        _run_synapses_2_post_initialise_queue();
        _run_synapses_4_post_initialise_queue();
        _run_synapses_5_post_initialise_queue();
        _run_synapses_post_initialise_queue();
        
                                    dev_dynamic_array_synapses_5__synaptic_pre.clear();
                                    dev_dynamic_array_synapses_5__synaptic_pre.shrink_to_fit();
                                    
        
                                    dev_dynamic_array_synapses__synaptic_pre.clear();
                                    dev_dynamic_array_synapses__synaptic_pre.shrink_to_fit();
                                    
        
                                    dev_dynamic_array_synapses_6__synaptic_pre.clear();
                                    dev_dynamic_array_synapses_6__synaptic_pre.shrink_to_fit();
                                    
        
                                    dev_dynamic_array_synapses_4__synaptic_pre.clear();
                                    dev_dynamic_array_synapses_4__synaptic_pre.shrink_to_fit();
                                    
        
                                    dev_dynamic_array_synapses_2__synaptic_pre.clear();
                                    dev_dynamic_array_synapses_2__synaptic_pre.shrink_to_fit();
                                    
        
                                    dev_dynamic_array_synapses_3__synaptic_pre.clear();
                                    dev_dynamic_array_synapses_3__synaptic_pre.shrink_to_fit();
                                    
        
                                    dev_dynamic_array_synapses_1__synaptic_pre.clear();
                                    dev_dynamic_array_synapses_1__synaptic_pre.shrink_to_fit();
                                    
        magicnetwork.clear();
        magicnetwork.add(&defaultclock, _run_random_number_buffer);
        magicnetwork.add(&defaultclock, _run_statemonitor_codeobject);
        magicnetwork.add(&defaultclock, _run_statemonitor_1_codeobject);
        magicnetwork.add(&defaultclock, _run_statemonitor_2_codeobject);
        magicnetwork.add(&defaultclock, _run_statemonitor_3_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_1_stateupdater_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_2_stateupdater_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_3_stateupdater_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_stateupdater_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_1_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_2_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_3_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_poissongroup_1_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_poissongroup_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_spikemonitor_codeobject);
        magicnetwork.add(&defaultclock, _run_spikemonitor_1_codeobject);
        magicnetwork.add(&defaultclock, _run_spikemonitor_2_codeobject);
        magicnetwork.add(&defaultclock, _run_spikemonitor_3_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_1_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_1_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_2_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_2_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_3_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_3_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_4_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_4_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_5_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_5_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_6_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_6_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_1_post_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_1_post_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_2_post_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_2_post_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_4_post_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_4_post_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_5_post_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_5_post_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_post_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_post_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_1_resetter_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_2_resetter_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_3_resetter_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_resetter_codeobject);
        magicnetwork.add(&defaultclock, _run_ratemonitor_codeobject);
        magicnetwork.add(&defaultclock, _run_ratemonitor_1_codeobject);
        magicnetwork.add(&defaultclock, _run_ratemonitor_2_codeobject);
        magicnetwork.add(&defaultclock, _run_ratemonitor_3_codeobject);
        CUDA_SAFE_CALL(hipProfilerStart());
        magicnetwork.run(2.0, NULL, 10.0);
        random_number_buffer.run_finished();
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        CUDA_SAFE_CALL(hipProfilerStop());
        _debugmsg_synapses_5_post_codeobject();
        
        _debugmsg_synapses_pre_codeobject();
        
        _copyToHost_spikemonitor_3_codeobject();
        _debugmsg_spikemonitor_3_codeobject();
        
        _debugmsg_synapses_5_pre_codeobject();
        
        _debugmsg_synapses_3_pre_codeobject();
        
        _debugmsg_synapses_4_pre_codeobject();
        
        _debugmsg_synapses_2_post_codeobject();
        
        _debugmsg_synapses_post_codeobject();
        
        _debugmsg_synapses_6_pre_codeobject();
        
        _copyToHost_spikemonitor_1_codeobject();
        _debugmsg_spikemonitor_1_codeobject();
        
        _debugmsg_synapses_4_post_codeobject();
        
        _debugmsg_synapses_2_pre_codeobject();
        
        _copyToHost_spikemonitor_codeobject();
        _debugmsg_spikemonitor_codeobject();
        
        _debugmsg_synapses_1_post_codeobject();
        
        _copyToHost_spikemonitor_2_codeobject();
        _debugmsg_spikemonitor_2_codeobject();
        
        _debugmsg_synapses_1_pre_codeobject();

    }

    const double _run_time3 = (double)(std::clock() -_start_time3)/CLOCKS_PER_SEC;
    printf("INFO: main_lines took %f seconds\n", _run_time3);

    brian_end();

    // Profiling
    const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    printf("INFO: main function took %f seconds\n", _run_time);

    return 0;
}